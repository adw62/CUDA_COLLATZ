#include "hip/hip_runtime.h"

#include <stdio.h>
#include <cassert>

__global__ void henon_kernel(double *xd_points, double *yd_points, double a, double b, int steps) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform Henon map computation
    double x = xd_points[idx];
    double y = yd_points[idx];
    for (int i = 0; i < steps; ++i) {
        double x_new = 1 - a * x * x + y;
        double y_new = b * x;
        x = x_new;
        y = y_new;
    }
    xd_points[idx] = x;
    yd_points[idx] = y;
}

__global__ void tinkerbell_kernel(double *xd_points, double *yd_points, double a, double b, double c, double d, int steps) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Define bounding box parameters
    double xmin = -200.0;
    double xmax = 200.0;
    double ymin = -200.0;
    double ymax = 200.0;

    // Perform Tinkerbell map computation
    double x = xd_points[idx];
    double y = yd_points[idx];
    for (int i = 0; i < steps; ++i) {
        double x_new = x * x - y * y  + a * x + b * y;
        double y_new = 2 * x * y + c * x + d * y;
        
        // Check if values exceed threshold
        if (fabs(x_new) > 1000000000 || fabs(y_new) > 1000000000) {
            // Set output values to 0 and return
            xd_points[idx] = 0.0;
            yd_points[idx] = 0.0;
            return;
        }

        x = x_new;
        y = y_new;
    }
    
    // Check if final values fall within bounding box
    if (x < xmin || x > xmax || y < ymin || y > ymax) {
        xd_points[idx] = 0.0;
        yd_points[idx] = 0.0;
    } else {
        xd_points[idx] = x;
        yd_points[idx] = y;
    }
}

__global__ void bogdanov_kernel(double *xd_points, double *yd_points, double eps, double k, double mew, int steps) {
    // 2D log map using doubles
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    double xn = xd_points[idx];
    double yn = yd_points[idx];
    double xn1 = xd_points[idx];
    double yn1 = yd_points[idx];
    for (int s = 0; s < steps; ++s) {
        yn1 = yn + eps * yn + k * xn * (xn - 1) + mew * xn * yn;
        xn1 = xn + yn1;

        xn = xn1;
        yn = yn1;
    }
    xd_points[idx] = xn;
    yd_points[idx] = yn;
}

__global__
void logistics_kernel(double *xd_points, double r, int steps) {
    //1D log map using doubles
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double xn = xd_points[idx];
    double xn1 = xd_points[idx];
    for (int s = 0; s<steps; ++s) {
    xn1 = r*xn*(1-xn);
    xn = xn1;
    }
    xd_points[idx] = xn;
}

__global__ void coll_kernel(int *a, int *b, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int res = a[idx];
        int count = 1;
        while (res != 1 && count < 10000) {
            count++;
            res = (res % 2 == 0) ? res / 2 : 3 * res + 1;
        }
        b[idx] = (res == 1) ? count : 888888;
    }
}

void collatz(int *start_ints, int N) {
    int *d_start_ints, *d_result_ints;

    hipMalloc((void **)&d_start_ints, N * sizeof(int));
    hipMalloc((void **)&d_result_ints, N * sizeof(int));

    hipMemcpy(d_start_ints, start_ints, N * sizeof(int), hipMemcpyHostToDevice);

    // Configure kernel launch parameters
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    coll_kernel<<<gridSize, blockSize>>>(d_start_ints, d_result_ints, N);

    hipMemcpy(start_ints, d_result_ints, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_start_ints);
    hipFree(d_result_ints);
}

void bogdanov(double *x_points, int N, double *y_points, int M, double eps, double k,
 double mew, int steps) {
    assert(N==M);

    double* xd_points;
    double* yd_points;

    hipMalloc((void **)&xd_points, N*sizeof(double));
    hipMalloc((void **)&yd_points, M*sizeof(double));

    hipMemcpy(xd_points, x_points, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(yd_points, y_points, M*sizeof(double), hipMemcpyHostToDevice);
    
    // Configure kernel launch parameters
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    bogdanov_kernel<<<gridSize, blockSize>>>(xd_points, yd_points, eps, k, mew, steps);

    hipMemcpy(x_points, xd_points, N*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y_points, yd_points, N*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(xd_points);
    hipFree(yd_points);
}

void tinkerbell(double *x_points, int N, double *y_points, int M, double a, double b, double c, double d, int steps) {
    assert(N == M);
    
    double* xd_points;
    double* yd_points;

    hipMalloc((void **)&xd_points, N * sizeof(double));
    hipMalloc((void **)&yd_points, M * sizeof(double));

    hipMemcpy(xd_points, x_points, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(yd_points, y_points, M * sizeof(double), hipMemcpyHostToDevice);

    // Configure kernel launch parameters
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    tinkerbell_kernel<<<gridSize, blockSize>>>(xd_points, yd_points, a, b, c, d, steps);

    hipMemcpy(x_points, xd_points, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y_points, yd_points, M * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(xd_points);
    hipFree(yd_points);
}

void logistics_map(double *x_points, int N, double r, int steps) {

    double* xd_points;

    hipMalloc((void **)&xd_points, N*sizeof(double));
    
    hipMemcpy(xd_points, x_points, N*sizeof(double), hipMemcpyHostToDevice);
    
    // Configure kernel launch parameters
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    logistics_kernel<<<gridSize, blockSize>>>(xd_points, r, steps);

    hipMemcpy(x_points, xd_points, N*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(xd_points);
}

void henon(double *x_points, int N, double *y_points, int M, double a, double b, int steps) {
    assert(N == M);

    double* xd_points;
    double* yd_points;

    hipMalloc((void **)&xd_points, N * sizeof(double));
    hipMalloc((void **)&yd_points, M * sizeof(double));

    hipMemcpy(xd_points, x_points, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(yd_points, y_points, M * sizeof(double), hipMemcpyHostToDevice);

    // Configure kernel launch parameters
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    henon_kernel<<<gridSize, blockSize>>>(xd_points, yd_points, a, b, steps);

    hipMemcpy(x_points, xd_points, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y_points, yd_points, M * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(xd_points);
    hipFree(yd_points);
}


